
#include <hip/hip_runtime.h>
#include <stdlib.h>			
#include <iostream>
#include <math.h>
#include <time.h>
#include <sys/time.h>

timeval t1, t2;

__global__ void Submanifold_conv(float* image, float* filter, float* result, int image_Rows, int image_Cols, int filterRC, int filter_Depth, int result_Rows, int result_Cols, int padding)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0;

	if (row < result_Rows && col < result_Cols)
	{       //printf("row %d col %d\n",row,col);
		int image_Row_Cols = image_Rows * image_Cols;

		for (int filterRow = 0; filterRow < filterRC; filterRow++) 
                {
			for (int filterCol = 0; filterCol < filterRC; filterCol++)
                        {
	                   for (int dep = 0; dep < filter_Depth; dep++)
                           {
                    
			    sum += image[(row + filterRow) * image_Cols + col + filterCol + dep * image_Row_Cols] * filter[filterRow * filterRC + filterCol + dep * filter_Depth];
                           }
			}
		}
                if(image[(row+padding) * (result_Cols+(padding*2)) + (col+padding)]!=0)
		result[row * result_Cols + col] = sum;
	}
}


void Convolution3D( float* image,  float* filter,  float* result, int padding, int image_Rows, int image_Cols, int filter_Rows, int filter_Depth,int result_Rows, int result_Cols)
{	int threadsPerBlock =32;

	int grid_Cols = ceil(float(result_Cols) / float(threadsPerBlock));
	int grid_Rows = ceil(float(result_Rows) / float(threadsPerBlock));

	dim3 gridDim(grid_Cols, grid_Rows);
	dim3 blockDim(threadsPerBlock,threadsPerBlock);		// total 32*32 = 1024 threads

        Submanifold_conv <<< gridDim, blockDim >>>(image,filter,result,image_Rows, image_Cols,filter_Rows, filter_Depth,result_Rows,result_Cols,padding );
}


int main() {

       	float *Mat1;//image
       	float *Mat2;//filter
        float *Mat3;//result
        float *padded_Mat1; 
	int filter_Size = 3;
        int padding=(filter_Size-1)/2;

	int Mat1_Rows = 4;
	int Mat1_Cols = 4;
	int Mat1_Depth = 3;
        int padded_Mat1_Rows = Mat1_Rows+(padding*2);
        int padded_Mat1_Cols = Mat1_Cols+(padding*2);
        int padded_Mat1_Depth = Mat1_Depth;

	int Mat2_Rows = filter_Size;
	int Mat2_Cols = filter_Size;
	int Mat2_Depth = 3;

	int Mat3_Rows = padded_Mat1_Rows - filter_Size + 1;
	int Mat3_Cols = padded_Mat1_Cols - filter_Size + 1;
	int Mat3_Depth = 1;

	int Mat1_Size = Mat1_Rows * Mat1_Cols * Mat1_Depth;
	int Mat2_Size = Mat2_Rows * Mat2_Cols * Mat2_Depth;
	int Mat3_Size = Mat3_Rows * Mat3_Cols * Mat3_Depth;
        int padded_Mat1_Size = padded_Mat1_Rows * padded_Mat1_Cols * padded_Mat1_Depth;
        
        //memory allocation
        hipMallocManaged( & Mat1, Mat1_Size *sizeof(float)); //places variables in unified memory, available to CPU and GPU
        hipMallocManaged( & Mat2, Mat2_Size *sizeof(float));
        hipMallocManaged( & Mat3, Mat3_Size *sizeof(float));
        hipMallocManaged( & padded_Mat1, padded_Mat1_Size *sizeof(float));
        int ii,jj,kk;

	for (int k = 0; k < padded_Mat1_Depth; k++)
        {

           for (int j=0; j < padded_Mat1_Cols; j++)
           {
	      for (int i=0; i < padded_Mat1_Rows; i++)
              {  
                   if((i==0 && j==0) || (i==2 && j ==0) || (i==1 && j==2))

                      { 
                           ii=i+padding;
                           jj=j+padding;
                           kk=k;    
                           padded_Mat1[(kk *padded_Mat1_Rows * padded_Mat1_Cols) + (jj * padded_Mat1_Rows) + ii]=1;
                      }
               
              }
           } 
        }
        
        for (size_t i = 0; i <Mat2_Size; i++)
           Mat2[i]=1;

        std::cout <<"padded_Mat1"<<std::endl;
        for (size_t dep = 0; dep < padded_Mat1_Depth; dep++)
        {  
          std::cout << std::endl;

          for (size_t col = 0; col < padded_Mat1_Cols; col++)
          {
                 for (size_t row = 0; row < padded_Mat1_Rows; row++)

                 {
                         std::cout << padded_Mat1[(dep *padded_Mat1_Rows * padded_Mat1_Cols) + (col * padded_Mat1_Rows) + row] << " ";
                 }
                 std::cout << std::endl;
          }
        }

        gettimeofday(&t1, 0); //time
	Convolution3D(padded_Mat1, Mat2, Mat3,padding,padded_Mat1_Rows,padded_Mat1_Cols, Mat2_Rows, Mat2_Depth,Mat3_Rows,Mat3_Cols);

	hipDeviceSynchronize();
        gettimeofday(&t2, 0);//time
        double time = t2.tv_sec+(t2.tv_usec/1000000.0)- t1.tv_sec-(t1.tv_usec/1000000.0);

        printf("Convolution time:  %.6lf  s \n", time);

	for (size_t row = 0; row < Mat3_Rows; row++)
	{
		for (size_t col = 0; col < Mat3_Cols; col++)
		{
			std::cout << Mat3[row * Mat3_Cols + col] << " ";
		}
		std::cout << std::endl;
	}
	

	// cpu and gpu memory free
	hipFree(Mat1);
	hipFree(Mat2);
	hipFree(Mat3);
        hipFree(padded_Mat1);

	return 0;
}
